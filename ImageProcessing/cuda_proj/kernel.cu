#include "hip/hip_runtime.h"
// How to "wrap" a CUDA kernel with a C++ class; the kernel must be defined outside of
// the class and launched from within a class instance's method.

#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define LEN 10

__global__ void kernel(int* a, int* b, unsigned int N);

class MyClass {
public:
	MyClass() {
		hipMalloc((void**)&data, sizeof(int) * LEN);
		hipMemset((void*)data, 0, sizeof(int) * LEN);
	};

	~MyClass() {
		hipFree((void*)data);
	};

	void run(int* b) {
		dim3 grid(1);
		dim3 block(LEN);
		kernel << <grid, block >> > (data, b, LEN);
	};

	int* get(void) {
		return data;
	};
private:
	int* data;
};

__global__ void kernel(int* a, int* b, unsigned int N) {
	const unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < N) {
		a[i] += b[i];
	}
}

void show(int* data, unsigned int N) {
	for (int i = 0; i < N; i++) {
		std::cout << data[i] << " ";
	}
	std::cout << std::endl;
}

int main(void) {
	MyClass c;
	int* b_gpu, b_host[LEN];

	for (int i = 0; i < LEN; i++) {
		b_host[i] = i;
	}

	hipMalloc((void**)&b_gpu, sizeof(int) * LEN);
	hipMemcpy(b_gpu, b_host, sizeof(int) * LEN, hipMemcpyHostToDevice);
	c.run(b_gpu);
	hipMemcpy(b_host, c.get(), sizeof(int) * LEN, hipMemcpyDeviceToHost);
	hipFree(b_gpu);

	show(b_host, LEN);
}