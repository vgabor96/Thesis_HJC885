#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <math.h>  
#include <stdio.h>
#include <stdlib.h> 
#include <time.h> 
#include "Timing.h"
#include <opencv2\core\mat.hpp>
#include <opencv2\core\cuda.inl.hpp>
#include <iostream>
#include <opencv2\highgui.hpp>
#include <opencv2\imgproc.hpp>
using namespace std;
using namespace cv;


const int N = 768;
const int M = 1024;

const int NM = N * M;

//Konstansok
const int RND_Min = 0;
const int RND_Max = 255;
const int MaxThreads = 1024;

__device__ const int dev_N = 768;
__device__ const int dev_M = 1024;
__device__ const int dev_NM = dev_N * dev_M;

__device__ const int dev_MaxThreads = 1024;
__device__ const int dev_Max = 255;

//M�r�si eszk�z�k:

hipEvent_t start;
hipEvent_t stop;

//RGB csatorn�k (eredeti k�p):
float channel_r[N][M];
float channel_g[N][M];
float channel_b[N][M];

//M�dos�tott k�pek:
int grayScale[N][M];     // fekete-feh�r k�p

int forMinMaxSearch[NM]; // vektor a Minimum �s Maximum keres�shez

int histogram[N][M];     // histogram korrekci�

int noNoise[N][M];		 // Gauss ut�n

int blackAndWhite[N][M]; // csak 0 �s 255 �rt�k� pixelek

int valueMatrix[N][M];	 // csak -3 �s 255 �rt�k� pixelek

//Seg�d v�ltoz�k:
int globalMin[1];		  // glob�lis Minimum
int globalMax[1];	      // glob�lis Maximum

int darkPixelCounter[1] = { 0 };

int avgPixelColor[1] = { 0 };
int avgPixelColorCounter[1] = { 0 };

//GAUSS M�trixok:
int GaussSize = 3;
int GaussValue[1] = { 0 };
int GaussMatrix[3][3] =
{
	{1, 2, 1},
	{2, 4, 2},
	{1, 2, 1}
};


//***********************************************************************************//

//GPU - RGB csatorn�k (eredeti k�p):
__device__ float dev_channel_r[N][M];
__device__ float dev_channel_g[N][M];
__device__ float dev_channel_b[N][M];

//GPU - M�dos�tott k�pek:
__device__ int dev_grayScale[N][M];     // fekete-feh�r k�p

__device__ int dev_forMinMaxSearch[NM]; // vektor a Minimum �s Maximum keres�shez

__device__ int dev_histogram[N][M];     // histogram korrekci�

__device__ int dev_noNoise[N][M];

__device__ int dev_blackAndWhite[N][M];

__device__ int dev_valueMatrix[N][M];


int res_grayScale[N][M];

int res_forMinMaxSearch[NM];

int res_histogram[N][M];

int res_noNoise[N][M];

int res_blackAndWhite[N][M];

int res_valueMatrix[N][M];

//GPU - Seg�d v�ltoz�k:
__device__ int localMin[(NM + 1) / 2];
__device__ int localMax[(NM + 1) / 2];

__device__ int dev_globalMin[1];		  // glob�lis Minimum
__device__ int dev_globalMax[1];	      // glob�lis Maximum

__device__ int dev_darkPixelCounter[1] = { 0 };

__device__ int dev_avgPixelColor[1] = { 0 };
__device__ int dev_avgPixelColorCounter[1] = { 0 };


int res_globalMin[1];		  // glob�lis Minimum
int res_globalMax[1];

int res_darkPixelCounter[1] = { 0 };

int res_avgPixelColor[1] = { 0 };

//GPU - GAUSS M�trixok:
__device__ int dev_GaussSize = 3;
__device__ int dev_GaussValue[1] = { 0 };
__device__ int dev_GaussMatrix[3][3] =
{
	{1, 2, 1},
	{2, 4, 2},
	{1, 2, 1}
};

int res_GaussValue[1] = { 0 };

//********************************************************************************************************************//
//Random k�p gener�l�sa:

void RandomPicture_CPU()
{
	srand(time(NULL));
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < M; j++)
		{
			channel_r[i][j] = (rand() % (RND_Min - RND_Max + 1)) + RND_Min;
			channel_g[i][j] = (rand() % (RND_Min - RND_Max + 1)) + RND_Min;
			channel_b[i][j] = (rand() % (RND_Min - RND_Max + 1)) + RND_Min;
		}
	}
}

//********************************************************************************************************************//
//K�p beolvas�sa:


//********************************************************************************************************************//
//Fekete feh�r:

__global__ void GrayScale()
{
	int j = threadIdx.x;

	if (dev_M <= dev_MaxThreads)
	{
		if (j < dev_M)
		{
			for (int i = 0; i < dev_N; i++)
			{
				dev_grayScale[i][j] = __float2int_rn((dev_channel_r[i][j] + dev_channel_g[i][j] + dev_channel_b[i][j]) / 3);
			}
		}
	}
	else
	{
		int run = __float2int_ru(dev_M / dev_MaxThreads);
		for (int block_i = 0; block_i < run; block_i++)
		{
			int modifier = block_i * dev_MaxThreads;

			for (int i = 0; i < N; i++)
			{
				if ((j + modifier) < dev_M)
				{
					dev_grayScale[i][j + modifier] = __float2int_rn((dev_channel_r[i][j + modifier] + dev_channel_g[i][j + modifier] + dev_channel_b[i][j + modifier]) / 3);
				}
			}
		}
	}
}

void GrayScale_CPU()
{
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < M; j++)
		{
			grayScale[i][j] = round((channel_r[i][j] + channel_g[i][j] + channel_b[i][j]) / 3);
		}
	}
}

//********************************************************************************************************************//
//2D t�mb --> 1D vektor konvert�l�s:

__global__ void ConvertArrayToVector()
{
	int j = threadIdx.x;

	if (dev_M <= dev_MaxThreads)
	{
		if (j < dev_M)
		{
			for (int i = 0; i < dev_N; i++)
			{
				int idx = (i * dev_M) + j;
				dev_forMinMaxSearch[idx] = dev_grayScale[i][j];
			}
		}
	}
	else
	{
		int run = __float2int_ru(dev_M / dev_MaxThreads);
		for (int block_i = 0; block_i < run; block_i++)
		{
			int modifier = block_i * dev_MaxThreads;
			for (int i = 0; i < dev_N; i++)
			{
				if (j + modifier < dev_M)
				{
					int idx = (i * dev_M) + j + modifier;
					dev_forMinMaxSearch[idx] = dev_grayScale[i][j + modifier];
				}
			}
		}
	}
}

//Minimum kiv�laszt�s:

__global__ void MinSearch()
{
	int blockSize = dev_NM;

	int num_1_idx = threadIdx.x * 2;
	int num_2_idx = threadIdx.x * 2 + 1;
	int localMinValue = -1;

	if (dev_M <= dev_MaxThreads)
	{
		if (threadIdx.x < (dev_NM + 1) / 2)
		{
			while (blockSize > 0)
			{
				if (blockSize == dev_NM)
				{
					if (num_1_idx < blockSize && num_2_idx < blockSize)
					{
						localMinValue = dev_forMinMaxSearch[num_1_idx] < dev_forMinMaxSearch[num_2_idx] ? dev_forMinMaxSearch[num_1_idx] : dev_forMinMaxSearch[num_2_idx];
					}

					if (num_1_idx < blockSize && blockSize <= num_2_idx)
					{
						localMinValue = dev_forMinMaxSearch[num_1_idx];
					}
					__syncthreads();

					localMin[threadIdx.x] = localMinValue;
					__syncthreads();
				}
				else
				{
					if (num_1_idx < blockSize && num_2_idx < blockSize)
					{
						localMinValue = localMin[num_1_idx] < localMin[num_2_idx] ? localMin[num_1_idx] : localMin[num_2_idx];
					}

					if (num_1_idx < blockSize && blockSize <= num_2_idx)
					{
						localMinValue = localMin[num_1_idx];
					}
					__syncthreads();

					localMin[threadIdx.x] = localMinValue;
					__syncthreads();
				}

				if (blockSize % 2 == 1 && blockSize != 1)
				{
					blockSize++;
				}

				blockSize = blockSize / 2;
			}
		}
	}
	else
	{
		int run = __float2int_ru(dev_M / dev_MaxThreads);
		for (int block_i = 0; block_i < run; block_i++)
		{
			int modifier = block_i * dev_MaxThreads;
			num_1_idx = (threadIdx.x + modifier) * 2;
			num_2_idx = (threadIdx.x + modifier) * 2 + 1;
			while (blockSize > 0)
			{
				if (blockSize == dev_NM)
				{
					if (num_1_idx < blockSize && num_2_idx < blockSize)
					{
						localMinValue = dev_forMinMaxSearch[num_1_idx] < dev_forMinMaxSearch[num_2_idx] ? dev_forMinMaxSearch[num_1_idx] : dev_forMinMaxSearch[num_2_idx];
					}

					if (num_1_idx < blockSize && blockSize <= num_2_idx)
					{
						localMinValue = dev_forMinMaxSearch[num_1_idx];
					}
					__syncthreads();

					localMin[threadIdx.x] = localMinValue;
					__syncthreads();
				}
				else
				{
					if (num_1_idx < blockSize && num_2_idx < blockSize)
					{
						localMinValue = localMin[num_1_idx] < localMin[num_2_idx] ? localMin[num_1_idx] : localMin[num_2_idx];
					}

					if (num_1_idx < blockSize && blockSize <= num_2_idx)
					{
						localMinValue = localMin[num_1_idx];
					}
					__syncthreads();

					localMin[threadIdx.x] = localMinValue;
					__syncthreads();
				}

				if (blockSize % 2 == 1 && blockSize != 1)
				{
					blockSize++;
				}

				blockSize = blockSize / 2;
			}
		}
	}

	if (threadIdx.x == 0)
	{
		dev_globalMin[0] = localMin[0];
	}
}

void MinSearch_CPU()
{
	globalMin[0] = grayScale[0][0];

	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < M; j++)
		{
			if (grayScale[i][j] < globalMin[0])
			{
				globalMin[0] = grayScale[i][j];
			}
		}
	}
}

//Maximum kiv�laszt�s:

__global__ void MaxSearch()
{
	int blockSize = dev_NM;

	int num_1_idx = threadIdx.x * 2;
	int num_2_idx = threadIdx.x * 2 + 1;
	int localMaxValue = -1;

	if (dev_M <= dev_MaxThreads)
	{
		if (threadIdx.x < (dev_NM + 1) / 2)
		{
			while (blockSize > 0)
			{
				if (blockSize == dev_NM)
				{
					if (num_1_idx < blockSize && num_2_idx < blockSize)
					{
						localMaxValue = dev_forMinMaxSearch[num_1_idx] > dev_forMinMaxSearch[num_2_idx] ? dev_forMinMaxSearch[num_1_idx] : dev_forMinMaxSearch[num_2_idx];
					}

					if (num_1_idx < blockSize && blockSize <= num_2_idx)
					{
						localMaxValue = dev_forMinMaxSearch[num_1_idx];
					}
					__syncthreads();

					localMax[threadIdx.x] = localMaxValue;
					__syncthreads();
				}
				else
				{
					if (num_1_idx < blockSize && num_2_idx < blockSize)
					{
						localMaxValue = localMax[num_1_idx] > localMax[num_2_idx] ? localMax[num_1_idx] : localMax[num_2_idx];
					}

					if (num_1_idx < blockSize && blockSize <= num_2_idx)
					{
						localMaxValue = localMax[num_1_idx];
					}
					__syncthreads();

					localMax[threadIdx.x] = localMaxValue;
					__syncthreads();
				}

				if (blockSize % 2 == 1 && blockSize != 1)
				{
					blockSize++;
				}

				blockSize = blockSize / 2;
			}
		}
	}
	else
	{
		int run = __float2int_ru(dev_M / dev_MaxThreads);
		for (int block_i = 0; block_i < run; block_i++)
		{
			int modifier = block_i * dev_MaxThreads;
			num_1_idx = (threadIdx.x + modifier) * 2;
			num_2_idx = (threadIdx.x + modifier) * 2 + 1;

			while (blockSize > 0)
			{
				if (blockSize == dev_NM)
				{
					if (num_1_idx < blockSize && num_2_idx < blockSize)
					{
						localMaxValue = dev_forMinMaxSearch[num_1_idx] > dev_forMinMaxSearch[num_2_idx] ? dev_forMinMaxSearch[num_1_idx] : dev_forMinMaxSearch[num_2_idx];
					}

					if (num_1_idx < blockSize && blockSize <= num_2_idx)
					{
						localMaxValue = dev_forMinMaxSearch[num_1_idx];
					}
					__syncthreads();

					localMax[threadIdx.x] = localMaxValue;
					__syncthreads();
				}
				else
				{
					if (num_1_idx < blockSize && num_2_idx < blockSize)
					{
						localMaxValue = localMax[num_1_idx] > localMax[num_2_idx] ? localMax[num_1_idx] : localMax[num_2_idx];
					}

					if (num_1_idx < blockSize && blockSize <= num_2_idx)
					{
						localMaxValue = localMax[num_1_idx];
					}
					__syncthreads();

					localMax[threadIdx.x] = localMaxValue;
					__syncthreads();
				}

				if (blockSize % 2 == 1 && blockSize != 1)
				{
					blockSize++;
				}

				blockSize = blockSize / 2;
			}
		}
	}

	if (threadIdx.x == 0)
	{
		dev_globalMax[0] = localMax[0];
	}
}

void MaxSearch_CPU()
{
	globalMax[0] = grayScale[0][0];

	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < M; j++)
		{
			if (globalMax[0] < grayScale[i][j])
			{
				globalMax[0] = grayScale[i][j];
			}
		}
	}
}

//Histogram sz�th�z�s:

__global__ void HistogramCorrection()
{
	int j = threadIdx.x;
	if (dev_M <= dev_MaxThreads)
	{
		if (threadIdx.x < dev_M)
		{
			for (int i = 0; i < dev_N; i++)
			{
				dev_histogram[i][j] = (dev_Max * (dev_grayScale[i][j] - dev_globalMin[0]) / (dev_globalMax[0] - dev_globalMin[0]));
			}
		}
	}
	else
	{
		int run = __float2int_ru(dev_M / dev_MaxThreads);
		for (int block_i = 0; block_i < run; block_i++)
		{
			int modifier = block_i * dev_MaxThreads;
			if (j + modifier < dev_M)
			{
				for (int i = 0; i < dev_N; i++)
				{
					dev_histogram[i][j + modifier] = (dev_Max * (dev_grayScale[i][j + modifier] - dev_globalMin[0]) / (dev_globalMax[0] - dev_globalMin[0]));
				}
			}
		}
	}
}

void HistogramCorrection_CPU()
{
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < M; j++)
		{
			histogram[i][j] = (RND_Max * (grayScale[i][j] - globalMin[0]) / (globalMax[0] - globalMin[0]));
		}
	}
}

//********************************************************************************************************************/
//Sz�n invert�l�s (ha kell):

__global__ void DarkPixelNumber()
{
	int j = threadIdx.x;
	if (dev_M <= dev_MaxThreads)
	{
		if (j < dev_M)
		{
			for (int i = 0; i < dev_N; i++)
			{
				if (dev_histogram[i][j] <= 127)
				{
					atomicAdd(&dev_darkPixelCounter[0], 1);
				}
			}
		}
	}
	else
	{
		int run = __float2int_ru(dev_M / dev_MaxThreads);
		for (int block_i = 0; block_i < run; block_i++)
		{
			int modifier = block_i * dev_MaxThreads;

			if (j + modifier < dev_M)
			{
				for (int i = 0; i < dev_N; i++)
				{
					if (dev_histogram[i][j + modifier] <= 127)
					{
						atomicAdd(&dev_darkPixelCounter[0], 1);
					}
				}
			}
		}
	}
}

void DarkPixelNumber_CPU()
{
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < M; j++)
		{
			if (histogram[i][j] <= 127)
			{
				darkPixelCounter[0]++;
			}
		}
	}
}

__global__ void ColorInverter()
{
	int j = threadIdx.x;
	if (dev_M <= dev_MaxThreads)
	{
		if (j < dev_M)
		{
			for (int i = 0; i < N; i++)
			{
				dev_histogram[i][j] = 255 - dev_histogram[i][j];
			}
		}
	}
	else
	{
		int run = __float2int_ru(dev_M / dev_MaxThreads);
		for (int block_i = 0; block_i < run; block_i++)
		{
			int modifier = block_i * dev_MaxThreads;
			if (j + modifier < dev_M)
			{
				for (int i = 0; i < N; i++)
				{
					dev_histogram[i][j + modifier] = 255 - dev_histogram[i][j + modifier];
				}
			}
		}
	}
}

void ColorInverter_CPU()
{
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < M; j++)
		{
			histogram[i][j] = 255 - histogram[i][j];
		}
	}
}

//********************************************************************************************************************/
//Zajsz�r�s:

//Gauss oszt� kisz�m�t�sa:

__global__ void GetGaussValue()
{
	int i = threadIdx.x / dev_GaussSize;
	int j = threadIdx.x - (i * dev_GaussSize);

	atomicAdd(&dev_GaussValue[0], dev_GaussMatrix[i][j]);
}

void GetGaussValue_CPU()
{
	for (int i = 0; i < GaussSize; i++)
	{
		for (int j = 0; j < GaussSize; j++)
		{
			GaussValue[0] += GaussMatrix[i][j];
		}
	}
}

//Gauss M�trix - Zajsz�r�s:

__global__ void GaussTransformation()
{
	int j = threadIdx.x;
	int diff = dev_GaussSize / 2;

	if (dev_M <= dev_MaxThreads)
	{
		if (j < dev_M)
		{
			for (int i = 0; i < dev_N; i++)
			{
				int sum = 0;
				int gauss_i = 0;
				int gauss_j = 0;

				for (int img_i = i - diff; img_i < i + diff; img_i++)
				{
					gauss_j = 0;
					for (int img_j = j - diff; img_j < j + diff; img_j++)
					{
						int originalValue = 0;

						if (0 <= img_i && 0 <= img_j && img_i < dev_N && img_j < dev_M)
						{
							originalValue = dev_histogram[img_i][img_j];
						}

						sum += dev_GaussMatrix[gauss_i][gauss_j] * originalValue;
						gauss_j++;
					}
					gauss_i++;
				}

				dev_noNoise[i][j] = sum / dev_GaussValue[0];
			}
		}
	}
	else
	{
		int run = __float2int_ru(dev_M / dev_MaxThreads);
		for (int block_i = 0; block_i < run; block_i++)
		{
			int modifier = block_i * dev_MaxThreads;
			if (j + modifier < dev_M)
			{
				for (int i = 0; i < dev_N; i++)
				{
					int sum = 0;
					int gauss_i = 0;
					int gauss_j = 0;

					for (int img_i = i - diff; img_i < i + diff; img_i++)
					{
						gauss_j = 0;
						for (int img_j = j + modifier - diff; img_j < j + modifier + diff; img_j++)
						{
							int originalValue = 0;

							if (0 <= img_i && 0 <= img_j && img_i < dev_N && img_j < dev_M)
							{
								originalValue = dev_histogram[img_i][img_j];
							}

							sum += dev_GaussMatrix[gauss_i][gauss_j] * originalValue;
							gauss_j++;
						}
						gauss_i++;
					}

					dev_noNoise[i][j + modifier] = sum / dev_GaussValue[0];
				}
			}
		}
	}
}

void GaussTransformation_CPU()
{
	int diff = dev_GaussSize / 2;
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < M; j++)
		{
			int sum = 0;
			int gauss_i = 0;
			int gauss_j = 0;

			for (int img_i = i - diff; img_i < i + diff; img_i++)
			{
				gauss_j = 0;
				for (int img_j = j - diff; img_j < j + diff; img_j++)
				{
					int originalValue = 0;

					if (0 <= img_i && 0 <= img_j && img_i < dev_N && img_j < dev_M)
					{
						originalValue = histogram[img_i][img_j];
					}

					sum += GaussMatrix[gauss_i][gauss_j] * originalValue;
					gauss_j++;
				}
				gauss_i++;
			}

			noNoise[i][j] = sum / GaussValue[0];
		}
	}
}

//********************************************************************************************************************/
//To Black And White:

//�tlagos pixel �rt�k

__global__ void AVGPixelColor()
{
	int j = threadIdx.x;

	if (dev_M <= dev_MaxThreads)
	{
		if (j < dev_M)
		{
			for (int i = 0; i < dev_N; i++)
			{
				//256 * 0,8 = 204,8 
				if (dev_noNoise[i][j] <= 205)
				{
					atomicAdd(&dev_avgPixelColor[0], dev_noNoise[i][j]);
					atomicAdd(&dev_avgPixelColorCounter[0], 1);
					__syncthreads();
				}
			}
		}
	}
	else
	{
		int run = __float2int_ru(dev_M / dev_MaxThreads);
		for (int block_i = 0; block_i < run; block_i++)
		{
			int modifier = block_i * dev_MaxThreads;
			if (j + modifier < dev_M)
			{
				for (int i = 0; i < dev_N; i++)
				{
					//256 * 0,8 = 204,8 
					if (dev_noNoise[i][j + modifier] <= 205)
					{
						atomicAdd(&dev_avgPixelColor[0], dev_noNoise[i][j + modifier]);
						atomicAdd(&dev_avgPixelColorCounter[0], 1);
						__syncthreads();
					}
				}
			}
		}
	}
	__syncthreads();
	if (threadIdx.x == 0)
	{
		dev_avgPixelColor[0] = __float2int_rn(dev_avgPixelColor[0] / dev_avgPixelColorCounter[0]);
	}
}

void AVGPixelColor_CPU()
{
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < M; j++)
		{
			if (dev_noNoise[i][j] <= 205)
			{
				avgPixelColor[0] += noNoise[i][j];
				avgPixelColorCounter[0]++;
			}
		}
	}

	avgPixelColor[0] = round(avgPixelColor[0] / avgPixelColorCounter[0]);
}

//Konvert�l�s 0-ra �s 255-re

__global__ void ConvertToBlackAndWhite()
{
	int j = threadIdx.x;
	if (dev_M <= dev_MaxThreads)
	{
		if (j < dev_M)
		{
			for (int i = 0; i < dev_N; i++)
			{
				if (dev_noNoise[i][j] <= dev_avgPixelColor[0])
				{
					dev_blackAndWhite[i][j] = 0;
				}
				else
				{
					dev_blackAndWhite[i][j] = 255;
				}
			}
		}
	}
	else
	{
		int run = __float2int_ru(dev_M / dev_MaxThreads);
		for (int block_i = 0; block_i < run; block_i++)
		{
			int modifier = block_i * dev_MaxThreads;
			if (j + modifier < dev_M)
			{
				for (int i = 0; i < dev_N; i++)
				{
					if (dev_noNoise[i][j + modifier] <= dev_avgPixelColor[0])
					{
						dev_blackAndWhite[i][j + modifier] = 0;
					}
					else
					{
						dev_blackAndWhite[i][j + modifier] = 255;
					}
				}
			}
		}
	}
}

void ConvertToBlackAndWhite_CPU()
{
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < M; j++)
		{
			if (noNoise[i][j] <= avgPixelColor[0])
			{
				blackAndWhite[i][j] = 0;
			}
			else
			{
				blackAndWhite[i][j] = 255;
			}
		}
	}
}

//********************************************************************************************************************/
//Konvert�l�s �rt�km�trix�:

__global__ void ConvertToValueMatrix()
{
	int j = threadIdx.x;
	if (dev_M <= dev_MaxThreads)
	{
		if (j < dev_M)
		{
			for (int i = 0; i < dev_N; i++)
			{
				if (dev_blackAndWhite[i][j] == 0)
				{
					dev_valueMatrix[i][j] = -3;
				}
				else
				{
					dev_valueMatrix[i][j] = 0;
				}
			}
		}
	}
	else
	{
		int run = __float2int_ru(dev_M / dev_MaxThreads);
		for (int block_i = 0; block_i < run; block_i++)
		{
			int modifier = block_i * dev_MaxThreads;
			if (j + modifier < dev_M)
			{
				for (int i = 0; i < dev_N; i++)
				{
					if (dev_blackAndWhite[i][j + modifier] == 0)
					{
						dev_valueMatrix[i][j + modifier] = -3;
					}
					else
					{
						dev_valueMatrix[i][j + modifier] = 0;
					}
				}
			}
		}
	}
}

void ConvertToValueMatrix_CPU()
{
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < M; j++)
		{
			if (blackAndWhite[i][j] == 0)
			{
				valueMatrix[i][j] = -3;
			}
			else
			{
				valueMatrix[i][j] = 0;
			}
		}
	}
}

//********************************************************************************************************************/
//Text file l�trehoz�sa, gener�l�sa:


//********************************************************************************************************************//
//Kiirat�s:

void Console_WriteLine(float arr[N][M])
{
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < M; j++)
		{
			printf("%003.0f	", arr[i][j]);
		}
		printf("\n");
	}
	printf("\n");
}

void Console_WriteLine(int arr[N][M])
{
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < M; j++)
		{
			printf("%003d	", arr[i][j]);
		}
		printf("\n");
	}
	printf("\n");
}

void Console_WriteLine(int vector[NM])
{
	for (int i = 0; i < NM; i++)
	{
		printf("%003d	", vector[i]);
	}
	printf("\n\n");
}

void Console_WriteLine(int number)
{
	printf("%003d", number);
	printf("\n\n");
}

void Console_WriteLine(char text[])
{
	printf("%s\n", text);
}

void Console_WriteLine(char text[], int number)
{
	printf("%s %d\n", text, number);
}

void Console_WriteLine(char text[], float number)
{
	printf("%s %f\n", text, number);
}

void Console_WriteLine(bool isCPU, char functionName[])
{
	if (isCPU)
	{
		printf("[CPU] - %s\n", functionName);
	}
	else
	{
		printf("[GPU] - %s\n", functionName);
	}
}

void Console_WriteLine(bool isCPU, char functionName[], float time)
{
	if (isCPU)
	{
		printf("[CPU] - %s, time: %f millisec\n", functionName, time);
	}
	else
	{
		printf("[GPU] - %s, time: %f millisec\n", functionName, time);
	}
}


void ClockStart()
{
	hipEventRecord(start);
}

void ClockStop(float& milliseconds)
{
	milliseconds = 0;
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
}

//********************************************************************************************************************//
//Main:

int main()
{

	float v_threshold = 10;
	//load the image
	//Mat img = imread("lena.jpg");
	Mat img = imread("C:\\Users\\loahc\\Documents\\GitHub\\Thesis_HJC885\\Unity\\Thesis_HJC885\\Assets\\screenshots\\testpic\\screen_1024x768_2019-11-25_15-22-04_01.png");
	Mat img2 = imread("C:\\Users\\loahc\\Documents\\GitHub\\Thesis_HJC885\\Unity\\Thesis_HJC885\\Assets\\screenshots\\testpic\\screen_1024x768_2019-11-25_15-22-04_02.png");
	
	namedWindow("pic_1", WINDOW_NORMAL);
	resizeWindow("pic_1", img.cols, img.rows);
	imshow("pic_1", img);

	namedWindow("newpic", WINDOW_NORMAL);
	resizeWindow("newpic", img2.cols, img2.rows);
	imshow("newpic", img2);

	waitKey(0);

	//read height, width data
//	Kernel();

	int height = img.rows;
	int width = img.cols;




	//check whether image loaded is empty or not.
	if (img.empty())
	{
		cerr << "no image"; return -1;
	}

	//convert image from CV::MAT to float*.
	Mat dst;
	img.convertTo(dst, CV_32F);
	float* data = dst.ptr<float>();

	Mat dst2;
	img2.convertTo(dst2, CV_32F);
	float* data2 = dst2.ptr<float>();


		//sub

	for (int i = 0; i < (height * width) * 3; i += 3)
	{
		float r = abs((data[i] - data2[i]) / 2);
		float g = abs((data[i + 1] - data2[i + 1]) / 2);
		float b = abs((data[i + 2] - data2[i + 2]) / 2);

		//float avg = (r + g + b) / 3;
		data[i] = r;
		data[i + 1] = g;
		data[i + 2] = b;
	}


	//convert back the image from float* to CV::MAT.
	Mat dest(height, width, CV_32FC3, data);

	//print the image
	imwrite("subpic.jpg", dest);
	Mat img3 = imread("subpic.jpg");
	namedWindow("subpic", WINDOW_NORMAL);
	resizeWindow("subpic", img3.cols, img3.rows);
	imshow("subpic", img3);

		//Makewhite and black only
	for (int i = 0; i < (height * width) * 3; i += 3)
	{

		float r = data[i];
		float g = data[i + 1];
		float b = data[i + 2];
		if (r >= v_threshold || g >= v_threshold || b >= v_threshold)
		{
			data[i] = 255;
			data[i + 1] = 255;
			data[i + 2] = 255;
		}


	}
	Mat dest2(height, width, CV_32FC3, data);

	//print the image
	imwrite("blackandwhite.jpg", dest2);
	Mat img5 = imread("blackandwhite.png",1);
	Mat img4 = imread("blackandwhite.jpg");
	printf("%i", img5.type());

	
	
	namedWindow("blackandwhite", WINDOW_NORMAL);
	resizeWindow("blackandwhite", img4.cols, img4.rows);
	imshow("blackandwhite", img5);
	waitKey(0);
	
	Mat src_gray;
	src_gray.convertTo(img5, CV_8U);

		cvtColor(src_gray, img5, COLOR_RGB2GRAY);
	


	/// Convert it to gray
	//cvtColor(img5, src_gray, COLOR_RGB2GRAY);

	/// Reduce the noise so we avoid false circle detection
	GaussianBlur(src_gray, src_gray, Size(9, 9), 2, 2);

	vector<Vec3f> circles;

	HoughCircles(img5, circles, HOUGH_GRADIENT, 1, src_gray.rows / 8, 200, 100, 0, 0);



	if (circles.size() == 0)
	{
		printf("No circles");
			return(-1);
	}
	int radius;
	for (int i = 0; i < circles.size(); i++)
	{
		Point center1(cvRound(circles[i][0]), cvRound(circles[i][1]));
		radius = cvRound(circles[i][2]);
		circle(img5, center1, 3, Scalar(0, 255, 0), -1, 8, 0);
		circle(img5, center1, radius, Scalar(255, 0, 0), 3, 8, 0);
		if (i>0)
		{
			Point center1(cvRound(circles[i-1][0]), cvRound(circles[i-1][1]));
			Point center2(cvRound(circles[i][0]), cvRound(circles[i][1]));
	
			line(img5, center1, center2, Scalar(255, 0, 255), 3, 8, 0);
		}
	}

	namedWindow("CIRCLES", WINDOW_NORMAL);
	resizeWindow("CIRCLES", img5.cols, img5.rows);
	imshow("CIRCLES", img5);
	waitKey(0);

	//////////////////////////////////////////////INNEN START////////////////////////////////////////////////////////////
	hipEventCreate(&start);
	hipEventCreate(&stop);

	float milliseconds = 0;

	Console_WriteLine("Picture Size N: ", N);
	Console_WriteLine("Picture Size M: ", M);

	Console_WriteLine("");
	Console_WriteLine("");

	MEASURE_TIME(1, "RandomPicture_CPU", RandomPicture_CPU());


	ClockStart();
	hipMemcpyToSymbol(HIP_SYMBOL(dev_channel_r), channel_r, N * M * sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(dev_channel_g), channel_g, N * M * sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(dev_channel_b), channel_b, N * M * sizeof(float));
	ClockStop(milliseconds);
	Console_WriteLine(false, "Copy", milliseconds);

	ClockStart();
	GrayScale << < 1, MaxThreads >> > ();
	ClockStop(milliseconds);
	Console_WriteLine(false, "Grayscale", milliseconds);



	ClockStart();
	ConvertArrayToVector << < 1, MaxThreads >> > ();

	ClockStop(milliseconds);
	Console_WriteLine(false, "Convert Array To Vector", milliseconds);



	ClockStart();
	MinSearch << < 1, MaxThreads >> > ();

	ClockStop(milliseconds);
	Console_WriteLine(false, "Minimum Search", milliseconds);



	ClockStart();
	MaxSearch << < 1, MaxThreads >> > ();

	ClockStop(milliseconds);
	Console_WriteLine(false, "Maximum Search", milliseconds);



	ClockStart();
	HistogramCorrection << < 1, MaxThreads >> > ();

	ClockStop(milliseconds);
	Console_WriteLine(false, "Historam Correction", milliseconds);



	ClockStart();
	DarkPixelNumber << <1, MaxThreads >> > ();

	ClockStop(milliseconds);
	Console_WriteLine(false, "Dark Pixel Counter", milliseconds);

	hipMemcpyFromSymbol(res_darkPixelCounter, HIP_SYMBOL(dev_darkPixelCounter), 1 * sizeof(int));

	if (NM / 2 < res_darkPixelCounter[0])
	{
		ClockStart();
		ColorInverter << <1, MaxThreads >> > ();

		ClockStop(milliseconds);
		Console_WriteLine(false, "Color Inverter", milliseconds);
	}



	ClockStart();
	GetGaussValue << <1, GaussSize* GaussSize >> > ();

	ClockStop(milliseconds);
	Console_WriteLine(false, "Gauss Value", milliseconds);



	ClockStart();
	GaussTransformation << <1, MaxThreads >> > ();

	ClockStop(milliseconds);
	Console_WriteLine(false, "Gauss Transformation", milliseconds);



	ClockStart();
	AVGPixelColor << <1, MaxThreads >> > ();

	ClockStop(milliseconds);
	Console_WriteLine(false, "AVG Pixel Color", milliseconds);



	ClockStart();
	ConvertToBlackAndWhite << <1, MaxThreads >> > ();

	ClockStop(milliseconds);
	Console_WriteLine(false, "Convert To Black And White", milliseconds);



	ClockStart();
	ConvertToValueMatrix << <1, MaxThreads >> > ();

	ClockStop(milliseconds);
	Console_WriteLine(false, "Convert To Value Matrix", milliseconds);



	Console_WriteLine("");
	Console_WriteLine("All GPU Process Finished");
	Console_WriteLine("");



	ClockStart();
	hipMemcpyFromSymbol(res_grayScale, HIP_SYMBOL(dev_grayScale), N * M * sizeof(int));
	hipMemcpyFromSymbol(res_forMinMaxSearch, HIP_SYMBOL(dev_forMinMaxSearch), N * M * sizeof(int));
	hipMemcpyFromSymbol(res_globalMin, HIP_SYMBOL(dev_globalMin), 1 * sizeof(int));
	hipMemcpyFromSymbol(res_globalMax, HIP_SYMBOL(dev_globalMax), 1 * sizeof(int));
	hipMemcpyFromSymbol(res_histogram, HIP_SYMBOL(dev_histogram), N * M * sizeof(int));
	hipMemcpyFromSymbol(res_GaussValue, HIP_SYMBOL(dev_GaussValue), 1 * sizeof(int));
	hipMemcpyFromSymbol(res_noNoise, HIP_SYMBOL(dev_noNoise), N * M * sizeof(int));
	hipMemcpyFromSymbol(res_avgPixelColor, HIP_SYMBOL(dev_avgPixelColor), 1 * sizeof(int));
	hipMemcpyFromSymbol(res_blackAndWhite, HIP_SYMBOL(dev_blackAndWhite), N * M * sizeof(int));
	hipMemcpyFromSymbol(res_valueMatrix, HIP_SYMBOL(dev_valueMatrix), N * M * sizeof(int));

	ClockStop(milliseconds);
	Console_WriteLine(false, "All Value Copy Back", milliseconds);


	MEASURE_TIME(1, "GrayScale_CPU", GrayScale_CPU());
	MEASURE_TIME(1, "MinSearch_CPU", MinSearch_CPU());
	MEASURE_TIME(1, "MaxSearch_CPU", MaxSearch_CPU());
	MEASURE_TIME(1, "HistogramCorrection_CPU", HistogramCorrection_CPU());
	MEASURE_TIME(1, "DarkPixelNumber_CPU", DarkPixelNumber_CPU());

	if (NM / 2 < darkPixelCounter[0])
	{
		MEASURE_TIME(1, "ColorInverter_CPU", ColorInverter_CPU());
	}

	MEASURE_TIME(1, "GetGaussValue_CPU", GetGaussValue_CPU());
	MEASURE_TIME(1, "GaussTransformation_CPU", GaussTransformation_CPU());
	MEASURE_TIME(1, "AVGPixelColor_CPU", AVGPixelColor_CPU());
	MEASURE_TIME(1, "ConvertToBlackAndWhite_CPU", ConvertToBlackAndWhite_CPU());
	MEASURE_TIME(1, "ConvertToValueMatrix_CPU", ConvertToValueMatrix_CPU());

	/*
	Console_WriteLine(channel_r);

	Console_WriteLine(channel_g);

	Console_WriteLine(channel_b);

	Console_WriteLine(grayScale);

	Console_WriteLine(forMinMaxSearch);

	Console_WriteLine(globalMin[0]);

	Console_WriteLine(globalMax[0]);

	Console_WriteLine(histogram);

	Console_WriteLine(GaussValue[0]);

	Console_WriteLine(noNoise);

	Console_WriteLine(avgPixelColor[0]);

	Console_WriteLine(blackAndWhite);

	Console_WriteLine(valueMatrix);
	*/


	return 0;
}