#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <math.h>  
#include <io.h>
#include <iostream>
#include <fstream>
#include <stdio.h>
#include <stdlib.h> 
#include <time.h> 
#include "Timing.h"
#include <opencv2\core\mat.hpp>
#include <opencv2\core\cuda.inl.hpp>
#include <iostream>
#include <opencv2\highgui.hpp>
#include <opencv2\imgproc.hpp>
using namespace std;
using namespace cv;


const int N = 1080; //768
const int M = 1920; //1024

int main()
{

	float v_threshold = 10;

	/*String folderpath = "C:\\Users\\loahc\\Documents\\GitHub\\Thesis_HJC885\\Unity\\Thesis_HJC885\\Assets\\screenshots\\train\\*.png";*/
	String folderpath = "C:\\Users\\loahc\\Documents\\GitHub\\Thesis_HJC885\\Unity\\Thesis_HJC885\\Assets\\screenshots\\test\\*.png";
		vector<String> filenames;
	cv::glob(folderpath, filenames);
	int hitcounter = 0;
	int nothitcounter = 0;
	printf("Picture Size N: %d \n", N);
	printf("Picture Size M: %d \n", M);
	int falsstring = 0;
	
	ofstream myfile;
	myfile.open("results.txt");
	//myfile.open("tests.txt");
	
	
	for (size_t x = 0; x < filenames.size()-1; x+=2)
	{
		Mat img = imread(filenames[x]);
		Mat img2 = imread(filenames[x+1]);


		unsigned first = filenames[x].find("(");
		unsigned last = filenames[x].find(")");
		string nametoattach = filenames[x].substr(first, last - (first-1));

		
		////SHOW IMAGE

		//namedWindow("pic_1", WINDOW_NORMAL);
		//resizeWindow("pic_1", img.cols, img.rows);
		//imshow("pic_1", img);

		//namedWindow("pic_2", WINDOW_NORMAL);
		//resizeWindow("pic_2", img2.cols, img2.rows);
		//imshow("pic_2", img2);




		int height = img.rows;
		int width = img.cols;
		//convert image from CV::MAT to float*.
		Mat dst;
		img.convertTo(dst, CV_32F);
		float* data = dst.ptr<float>();

		Mat dst2;
		img2.convertTo(dst2, CV_32F);
		float* data2 = dst2.ptr<float>();

		//sub

		for (int i = 0; i < (height * width) * 3; i += 3)
		{
			float r = abs((data[i] - data2[i]) / 2);
			float g = abs((data[i + 1] - data2[i + 1]) / 2);
			float b = abs((data[i + 2] - data2[i + 2]) / 2);

			//float avg = (r + g + b) / 3;
			data[i] = r;
			data[i + 1] = g;
			data[i + 2] = b;
		}


		//convert back the image from float* to CV::MAT.
		Mat dest(height, width, CV_32FC3, data);

		//print the image
		imwrite("subpic.jpg", dest);
		Mat img3 = imread("subpic.jpg");

		//SHOW IAMGE
		namedWindow("subpic", WINDOW_NORMAL);
		resizeWindow("subpic", img3.cols, img3.rows);
		imshow("subpic", img3);


		//Makewhite and black only
		for (int i = 0; i < (height * width) * 3; i += 3)
		{

			float r = data[i];
			float g = data[i + 1];
			float b = data[i + 2];
			if (r >= v_threshold || g >= v_threshold || b >= v_threshold)
			{
				data[i] = 255;
				data[i + 1] = 255;
				data[i + 2] = 255;
			}


		}

		//convert back the image from float* to CV::MAT.
		Mat dest3(height, width, CV_32FC3, data);

		//print the image
		imwrite("pic1_black.jpg", dest3);
		Mat img4 = imread("pic1_black.jpg");
		//SHOW IMAGE
		//namedWindow("pic1_black", WINDOW_NORMAL);
		//resizeWindow("pic1_black", img4.cols, img4.rows);
		//imshow("pic1_black", img4);

		Mat src, src_gray;

		/// Read the image
		src = img4;

		if (!src.data)
		{
			return -1;
		}

		/// Convert it to gray
		cvtColor(src, src_gray, COLOR_RGB2GRAY);
		//SHOW IMAGE
		//namedWindow("Hough Circle Transform Gray", WINDOW_NORMAL);
		//imshow("Hough Circle Transform Gray", src);

		/// Reduce the noise so we avoid false circle detection
		GaussianBlur(src_gray, src_gray, Size(9,9), 2, 2); //Size(9, 9), 2, 2);
		////SHOW IMAGE
		//namedWindow("Hough Circle Transform Gauss", WINDOW_NORMAL);
		//imshow("Hough Circle Transform Gauss", src);

		vector<Vec3f> circles;
		

			/// Apply the Hough Transform to find the circles
			//		WorkingHoughCircles(src_gray, circles, HOUGH_GRADIENT, 1, src_gray.rows / 100, 10, 10, 0, 100);
			//HoughCircles(src_gray, circles, HOUGH_GRADIENT, 1,0.5, 10, 7, 0, 10);
			//HoughCircles(src_gray, circles, HOUGH_GRADIENT, 1,5, 10, 7, 1, 10);
		HoughCircles(src_gray, circles, HOUGH_GRADIENT, 1, src_gray.rows / 100, 10, 10, 0, 100);

			/// Draw the circles detected
			for (size_t i = 0; i < circles.size(); i++)
			{
				Point center(cvRound(circles[i][0]), cvRound(circles[i][1]));
				int radius = cvRound(circles[i][2]);
				// circle center
				circle(src, center, 2, Scalar(0, 255, 0), 2, 8, 0);
				// circle outline
				circle(src, center, radius, Scalar(0, 0, 255), 2, 8, 0);
				if (i > 0 && i< 2)
				{
					if (filenames[x].find("True") != std::string::npos) {
						falsstring = 1;
						hitcounter++;
					}
					else
					{
						falsstring = 0;
						nothitcounter++;
					}

					Point center1(cvRound(circles[i - 1][0]), cvRound(circles[i - 1][1]));
					printf("%i,", center1.x);
					printf("%i,", center1.y);
				

					Point center2(cvRound(circles[i][0]), cvRound(circles[i][1]));

					printf("%i,", center2.x);
					printf("%i,", center2.y);
					printf("%i\n",falsstring);
					myfile << nametoattach<< "\t" << center1.x << "\t" << center1.y <<"\t"<< center2.x << "\t" << center2.y<<"\t"<<falsstring<<"\n";
					line(src, center1, center2, Scalar(0, 0, 255), 2, 8, 0);
				}
			}
		


		/// Show your results
		namedWindow("Hough Circle Transform", WINDOW_NORMAL);
		imshow("Hough Circle Transform", src);

		//WAIT OR NOT
		waitKey(0);
	}



	printf("Number of Hits: %i\n", hitcounter);
	printf("Number of Non-Hits: %i\n", nothitcounter);
	myfile.close();
	return 0;
}