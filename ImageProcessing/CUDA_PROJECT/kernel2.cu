#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <math.h>  
#include <stdio.h>
#include <stdlib.h> 
#include <time.h> 
#include "Timing.h"
#include <opencv2\core\mat.hpp>
#include <opencv2\core\cuda.inl.hpp>
#include <iostream>
#include <opencv2\highgui.hpp>
#include <opencv2\imgproc.hpp>
using namespace std;
using namespace cv;


const int N = 768;
const int M = 1024;

const int NM = N * M;


const int RND_Min = 0;
const int RND_Max = 255;
const int MaxThreads = 1024;

__device__ const int dev_N = 768;
__device__ const int dev_M = 1024;
__device__ const int dev_NM = dev_N * dev_M;

__device__ const int dev_MaxThreads = 1024;
__device__ const int dev_Max = 255;


//Contributed by Pakos

hipEvent_t start;
hipEvent_t stop;


float channel_r[N][M];
float channel_g[N][M];
float channel_b[N][M];


int grayScale[N][M];    

int forMinMaxSearch[NM];

int histogram[N][M];    

int noNoise[N][M];		

int blackAndWhite[N][M]; 

int valueMatrix[N][M];	 


int globalMin[1];		  
int globalMax[1];	      

int darkPixelCounter[1] = { 0 };

int avgPixelColor[1] = { 0 };
int avgPixelColorCounter[1] = { 0 };


int GaussSize = 3;
int GaussValue[1] = { 0 };
int GaussMatrix[3][3] =
{
	{1, 2, 1},
	{2, 4, 2},
	{1, 2, 1}
};





__device__ float dev_channel_r[N][M];
__device__ float dev_channel_g[N][M];
__device__ float dev_channel_b[N][M];

__device__ int dev_grayScale[N][M];    

__device__ int dev_forMinMaxSearch[NM]; 

__device__ int dev_histogram[N][M];    

__device__ int dev_noNoise[N][M];

__device__ int dev_blackAndWhite[N][M];

__device__ int dev_valueMatrix[N][M];


int res_grayScale[N][M];

int res_forMinMaxSearch[NM];

int res_histogram[N][M];

int res_noNoise[N][M];

int res_blackAndWhite[N][M];

int res_valueMatrix[N][M];


__device__ int localMin[(NM + 1) / 2];
__device__ int localMax[(NM + 1) / 2];

__device__ int dev_globalMin[1];		  
__device__ int dev_globalMax[1];	     

__device__ int dev_darkPixelCounter[1] = { 0 };

__device__ int dev_avgPixelColor[1] = { 0 };
__device__ int dev_avgPixelColorCounter[1] = { 0 };


int res_globalMin[1];		
int res_globalMax[1];

int res_darkPixelCounter[1] = { 0 };

int res_avgPixelColor[1] = { 0 };


__device__ int dev_GaussSize = 3;
__device__ int dev_GaussValue[1] = { 0 };
__device__ int dev_GaussMatrix[3][3] =
{
	{1, 2, 1},
	{2, 4, 2},
	{1, 2, 1}
};

int res_GaussValue[1] = { 0 };


void RandomPicture_CPU()
{
	srand(time(NULL));
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < M; j++)
		{
			channel_r[i][j] = (rand() % (RND_Min - RND_Max + 1)) + RND_Min;
			channel_g[i][j] = (rand() % (RND_Min - RND_Max + 1)) + RND_Min;
			channel_b[i][j] = (rand() % (RND_Min - RND_Max + 1)) + RND_Min;
		}
	}
}


void LoadPicture_CPU(Mat img)
{
	Mat img2 = imread("C:\\Users\\loahc\\Documents\\GitHub\\Thesis_HJC885\\Unity\\Thesis_HJC885\\Assets\\screenshots\\testpic\\screen_1024x768_2019-11-25_15-22-04_01.png");
	Mat dst;
	img2.convertTo(dst, CV_32F);
	float* data = dst.ptr<float>();

	int x = 0;

	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < M; j++)
		{

			//channel_r[i][j] = data[i * M + j *3];
			channel_r[i][j] = data[x];
			channel_g[i][j] = data[x + 1];
			channel_b[i][j] = data[x + 2];
			x += 3;

		}
	
	
	}
								
}

Mat LoadBackPicture()
{
	Mat img,r,g,b;

	r = Mat(N, M, CV_32FC1, channel_r,0);
	g = Mat(N, M, CV_32FC1, channel_g,0);
	b = Mat(N, M, CV_32FC1, channel_b,0);

	
	vector<Mat> channels;
	

	channels.push_back(r);
	channels.push_back(g);
	channels.push_back(b);

	merge(channels, img);


	return img;

}

//********************************************************************************************************************//
//Fekete feh�r:

__global__ void GrayScale()
{
	int j = threadIdx.x;

	if (dev_M <= dev_MaxThreads)
	{
		if (j < dev_M)
		{
			for (int i = 0; i < dev_N; i++)
			{
				dev_grayScale[i][j] = __float2int_rn((dev_channel_r[i][j] + dev_channel_g[i][j] + dev_channel_b[i][j]) / 3);
			}
		}
	}
	else
	{
		int run = __float2int_ru(dev_M / dev_MaxThreads);
		for (int block_i = 0; block_i < run; block_i++)
		{
			int modifier = block_i * dev_MaxThreads;

			for (int i = 0; i < N; i++)
			{
				if ((j + modifier) < dev_M)
				{
					dev_grayScale[i][j + modifier] = __float2int_rn((dev_channel_r[i][j + modifier] + dev_channel_g[i][j + modifier] + dev_channel_b[i][j + modifier]) / 3);
				}
			}
		}
	}
}

void GrayScale_CPU()
{
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < M; j++)
		{
			grayScale[i][j] = round((channel_r[i][j] + channel_g[i][j] + channel_b[i][j]) / 3);
		}
	}
}


__global__ void ConvertArrayToVector()
{
	int j = threadIdx.x;

	if (dev_M <= dev_MaxThreads)
	{
		if (j < dev_M)
		{
			for (int i = 0; i < dev_N; i++)
			{
				int idx = (i * dev_M) + j;
				dev_forMinMaxSearch[idx] = dev_grayScale[i][j];
			}
		}
	}
	else
	{
		int run = __float2int_ru(dev_M / dev_MaxThreads);
		for (int block_i = 0; block_i < run; block_i++)
		{
			int modifier = block_i * dev_MaxThreads;
			for (int i = 0; i < dev_N; i++)
			{
				if (j + modifier < dev_M)
				{
					int idx = (i * dev_M) + j + modifier;
					dev_forMinMaxSearch[idx] = dev_grayScale[i][j + modifier];
				}
			}
		}
	}
}


__global__ void MinSearch()
{
	int blockSize = dev_NM;

	int num_1_idx = threadIdx.x * 2;
	int num_2_idx = threadIdx.x * 2 + 1;
	int localMinValue = -1;

	if (dev_M <= dev_MaxThreads)
	{
		if (threadIdx.x < (dev_NM + 1) / 2)
		{
			while (blockSize > 0)
			{
				if (blockSize == dev_NM)
				{
					if (num_1_idx < blockSize && num_2_idx < blockSize)
					{
						localMinValue = dev_forMinMaxSearch[num_1_idx] < dev_forMinMaxSearch[num_2_idx] ? dev_forMinMaxSearch[num_1_idx] : dev_forMinMaxSearch[num_2_idx];
					}

					if (num_1_idx < blockSize && blockSize <= num_2_idx)
					{
						localMinValue = dev_forMinMaxSearch[num_1_idx];
					}
					__syncthreads();

					localMin[threadIdx.x] = localMinValue;
					__syncthreads();
				}
				else
				{
					if (num_1_idx < blockSize && num_2_idx < blockSize)
					{
						localMinValue = localMin[num_1_idx] < localMin[num_2_idx] ? localMin[num_1_idx] : localMin[num_2_idx];
					}

					if (num_1_idx < blockSize && blockSize <= num_2_idx)
					{
						localMinValue = localMin[num_1_idx];
					}
					__syncthreads();

					localMin[threadIdx.x] = localMinValue;
					__syncthreads();
				}

				if (blockSize % 2 == 1 && blockSize != 1)
				{
					blockSize++;
				}

				blockSize = blockSize / 2;
			}
		}
	}
	else
	{
		int run = __float2int_ru(dev_M / dev_MaxThreads);
		for (int block_i = 0; block_i < run; block_i++)
		{
			int modifier = block_i * dev_MaxThreads;
			num_1_idx = (threadIdx.x + modifier) * 2;
			num_2_idx = (threadIdx.x + modifier) * 2 + 1;
			while (blockSize > 0)
			{
				if (blockSize == dev_NM)
				{
					if (num_1_idx < blockSize && num_2_idx < blockSize)
					{
						localMinValue = dev_forMinMaxSearch[num_1_idx] < dev_forMinMaxSearch[num_2_idx] ? dev_forMinMaxSearch[num_1_idx] : dev_forMinMaxSearch[num_2_idx];
					}

					if (num_1_idx < blockSize && blockSize <= num_2_idx)
					{
						localMinValue = dev_forMinMaxSearch[num_1_idx];
					}
					__syncthreads();

					localMin[threadIdx.x] = localMinValue;
					__syncthreads();
				}
				else
				{
					if (num_1_idx < blockSize && num_2_idx < blockSize)
					{
						localMinValue = localMin[num_1_idx] < localMin[num_2_idx] ? localMin[num_1_idx] : localMin[num_2_idx];
					}

					if (num_1_idx < blockSize && blockSize <= num_2_idx)
					{
						localMinValue = localMin[num_1_idx];
					}
					__syncthreads();

					localMin[threadIdx.x] = localMinValue;
					__syncthreads();
				}

				if (blockSize % 2 == 1 && blockSize != 1)
				{
					blockSize++;
				}

				blockSize = blockSize / 2;
			}
		}
	}

	if (threadIdx.x == 0)
	{
		dev_globalMin[0] = localMin[0];
	}
}

void MinSearch_CPU()
{
	globalMin[0] = grayScale[0][0];

	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < M; j++)
		{
			if (grayScale[i][j] < globalMin[0])
			{
				globalMin[0] = grayScale[i][j];
			}
		}
	}
}


__global__ void MaxSearch()
{
	int blockSize = dev_NM;

	int num_1_idx = threadIdx.x * 2;
	int num_2_idx = threadIdx.x * 2 + 1;
	int localMaxValue = -1;

	if (dev_M <= dev_MaxThreads)
	{
		if (threadIdx.x < (dev_NM + 1) / 2)
		{
			while (blockSize > 0)
			{
				if (blockSize == dev_NM)
				{
					if (num_1_idx < blockSize && num_2_idx < blockSize)
					{
						localMaxValue = dev_forMinMaxSearch[num_1_idx] > dev_forMinMaxSearch[num_2_idx] ? dev_forMinMaxSearch[num_1_idx] : dev_forMinMaxSearch[num_2_idx];
					}

					if (num_1_idx < blockSize && blockSize <= num_2_idx)
					{
						localMaxValue = dev_forMinMaxSearch[num_1_idx];
					}
					__syncthreads();

					localMax[threadIdx.x] = localMaxValue;
					__syncthreads();
				}
				else
				{
					if (num_1_idx < blockSize && num_2_idx < blockSize)
					{
						localMaxValue = localMax[num_1_idx] > localMax[num_2_idx] ? localMax[num_1_idx] : localMax[num_2_idx];
					}

					if (num_1_idx < blockSize && blockSize <= num_2_idx)
					{
						localMaxValue = localMax[num_1_idx];
					}
					__syncthreads();

					localMax[threadIdx.x] = localMaxValue;
					__syncthreads();
				}

				if (blockSize % 2 == 1 && blockSize != 1)
				{
					blockSize++;
				}

				blockSize = blockSize / 2;
			}
		}
	}
	else
	{
		int run = __float2int_ru(dev_M / dev_MaxThreads);
		for (int block_i = 0; block_i < run; block_i++)
		{
			int modifier = block_i * dev_MaxThreads;
			num_1_idx = (threadIdx.x + modifier) * 2;
			num_2_idx = (threadIdx.x + modifier) * 2 + 1;

			while (blockSize > 0)
			{
				if (blockSize == dev_NM)
				{
					if (num_1_idx < blockSize && num_2_idx < blockSize)
					{
						localMaxValue = dev_forMinMaxSearch[num_1_idx] > dev_forMinMaxSearch[num_2_idx] ? dev_forMinMaxSearch[num_1_idx] : dev_forMinMaxSearch[num_2_idx];
					}

					if (num_1_idx < blockSize && blockSize <= num_2_idx)
					{
						localMaxValue = dev_forMinMaxSearch[num_1_idx];
					}
					__syncthreads();

					localMax[threadIdx.x] = localMaxValue;
					__syncthreads();
				}
				else
				{
					if (num_1_idx < blockSize && num_2_idx < blockSize)
					{
						localMaxValue = localMax[num_1_idx] > localMax[num_2_idx] ? localMax[num_1_idx] : localMax[num_2_idx];
					}

					if (num_1_idx < blockSize && blockSize <= num_2_idx)
					{
						localMaxValue = localMax[num_1_idx];
					}
					__syncthreads();

					localMax[threadIdx.x] = localMaxValue;
					__syncthreads();
				}

				if (blockSize % 2 == 1 && blockSize != 1)
				{
					blockSize++;
				}

				blockSize = blockSize / 2;
			}
		}
	}

	if (threadIdx.x == 0)
	{
		dev_globalMax[0] = localMax[0];
	}
}

void MaxSearch_CPU()
{
	globalMax[0] = grayScale[0][0];

	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < M; j++)
		{
			if (globalMax[0] < grayScale[i][j])
			{
				globalMax[0] = grayScale[i][j];
			}
		}
	}
}


__global__ void HistogramCorrection()
{
	int j = threadIdx.x;
	if (dev_M <= dev_MaxThreads)
	{
		if (threadIdx.x < dev_M)
		{
			for (int i = 0; i < dev_N; i++)
			{
				dev_histogram[i][j] = (dev_Max * (dev_grayScale[i][j] - dev_globalMin[0]) / (dev_globalMax[0] - dev_globalMin[0]));
			}
		}
	}
	else
	{
		int run = __float2int_ru(dev_M / dev_MaxThreads);
		for (int block_i = 0; block_i < run; block_i++)
		{
			int modifier = block_i * dev_MaxThreads;
			if (j + modifier < dev_M)
			{
				for (int i = 0; i < dev_N; i++)
				{
					dev_histogram[i][j + modifier] = (dev_Max * (dev_grayScale[i][j + modifier] - dev_globalMin[0]) / (dev_globalMax[0] - dev_globalMin[0]));
				}
			}
		}
	}
}

void HistogramCorrection_CPU()
{
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < M; j++)
		{
			histogram[i][j] = (RND_Max * (grayScale[i][j] - globalMin[0]) / (globalMax[0] - globalMin[0]));
		}
	}
}


__global__ void DarkPixelNumber()
{
	int j = threadIdx.x;
	if (dev_M <= dev_MaxThreads)
	{
		if (j < dev_M)
		{
			for (int i = 0; i < dev_N; i++)
			{
				if (dev_histogram[i][j] <= 127)
				{
					atomicAdd(&dev_darkPixelCounter[0], 1);
				}
			}
		}
	}
	else
	{
		int run = __float2int_ru(dev_M / dev_MaxThreads);
		for (int block_i = 0; block_i < run; block_i++)
		{
			int modifier = block_i * dev_MaxThreads;

			if (j + modifier < dev_M)
			{
				for (int i = 0; i < dev_N; i++)
				{
					if (dev_histogram[i][j + modifier] <= 127)
					{
						atomicAdd(&dev_darkPixelCounter[0], 1);
					}
				}
			}
		}
	}
}

void DarkPixelNumber_CPU()
{
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < M; j++)
		{
			if (histogram[i][j] <= 127)
			{
				darkPixelCounter[0]++;
			}
		}
	}
}

__global__ void ColorInverter()
{
	int j = threadIdx.x;
	if (dev_M <= dev_MaxThreads)
	{
		if (j < dev_M)
		{
			for (int i = 0; i < N; i++)
			{
				dev_histogram[i][j] = 255 - dev_histogram[i][j];
			}
		}
	}
	else
	{
		int run = __float2int_ru(dev_M / dev_MaxThreads);
		for (int block_i = 0; block_i < run; block_i++)
		{
			int modifier = block_i * dev_MaxThreads;
			if (j + modifier < dev_M)
			{
				for (int i = 0; i < N; i++)
				{
					dev_histogram[i][j + modifier] = 255 - dev_histogram[i][j + modifier];
				}
			}
		}
	}
}

void ColorInverter_CPU()
{
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < M; j++)
		{
			histogram[i][j] = 255 - histogram[i][j];
		}
	}
}


__global__ void GetGaussValue()
{
	int i = threadIdx.x / dev_GaussSize;
	int j = threadIdx.x - (i * dev_GaussSize);

	atomicAdd(&dev_GaussValue[0], dev_GaussMatrix[i][j]);
}

void GetGaussValue_CPU()
{
	for (int i = 0; i < GaussSize; i++)
	{
		for (int j = 0; j < GaussSize; j++)
		{
			GaussValue[0] += GaussMatrix[i][j];
		}
	}
}


__global__ void GaussTransformation()
{
	int j = threadIdx.x;
	int diff = dev_GaussSize / 2;

	if (dev_M <= dev_MaxThreads)
	{
		if (j < dev_M)
		{
			for (int i = 0; i < dev_N; i++)
			{
				int sum = 0;
				int gauss_i = 0;
				int gauss_j = 0;

				for (int img_i = i - diff; img_i < i + diff; img_i++)
				{
					gauss_j = 0;
					for (int img_j = j - diff; img_j < j + diff; img_j++)
					{
						int originalValue = 0;

						if (0 <= img_i && 0 <= img_j && img_i < dev_N && img_j < dev_M)
						{
							originalValue = dev_histogram[img_i][img_j];
						}

						sum += dev_GaussMatrix[gauss_i][gauss_j] * originalValue;
						gauss_j++;
					}
					gauss_i++;
				}

				dev_noNoise[i][j] = sum / dev_GaussValue[0];
			}
		}
	}
	else
	{
		int run = __float2int_ru(dev_M / dev_MaxThreads);
		for (int block_i = 0; block_i < run; block_i++)
		{
			int modifier = block_i * dev_MaxThreads;
			if (j + modifier < dev_M)
			{
				for (int i = 0; i < dev_N; i++)
				{
					int sum = 0;
					int gauss_i = 0;
					int gauss_j = 0;

					for (int img_i = i - diff; img_i < i + diff; img_i++)
					{
						gauss_j = 0;
						for (int img_j = j + modifier - diff; img_j < j + modifier + diff; img_j++)
						{
							int originalValue = 0;

							if (0 <= img_i && 0 <= img_j && img_i < dev_N && img_j < dev_M)
							{
								originalValue = dev_histogram[img_i][img_j];
							}

							sum += dev_GaussMatrix[gauss_i][gauss_j] * originalValue;
							gauss_j++;
						}
						gauss_i++;
					}

					dev_noNoise[i][j + modifier] = sum / dev_GaussValue[0];
				}
			}
		}
	}
}

void GaussTransformation_CPU()
{
	int diff = dev_GaussSize / 2;
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < M; j++)
		{
			int sum = 0;
			int gauss_i = 0;
			int gauss_j = 0;

			for (int img_i = i - diff; img_i < i + diff; img_i++)
			{
				gauss_j = 0;
				for (int img_j = j - diff; img_j < j + diff; img_j++)
				{
					int originalValue = 0;

					if (0 <= img_i && 0 <= img_j && img_i < dev_N && img_j < dev_M)
					{
						originalValue = histogram[img_i][img_j];
					}

					sum += GaussMatrix[gauss_i][gauss_j] * originalValue;
					gauss_j++;
				}
				gauss_i++;
			}

			noNoise[i][j] = sum / GaussValue[0];
		}
	}
}



__global__ void AVGPixelColor()
{
	int j = threadIdx.x;

	if (dev_M <= dev_MaxThreads)
	{
		if (j < dev_M)
		{
			for (int i = 0; i < dev_N; i++)
			{
				if (dev_noNoise[i][j] <= 205)
				{
					atomicAdd(&dev_avgPixelColor[0], dev_noNoise[i][j]);
					atomicAdd(&dev_avgPixelColorCounter[0], 1);
					__syncthreads();
				}
			}
		}
	}
	else
	{
		int run = __float2int_ru(dev_M / dev_MaxThreads);
		for (int block_i = 0; block_i < run; block_i++)
		{
			int modifier = block_i * dev_MaxThreads;
			if (j + modifier < dev_M)
			{
				for (int i = 0; i < dev_N; i++)
				{
					//256 * 0,8 = 204,8 
					if (dev_noNoise[i][j + modifier] <= 205)
					{
						atomicAdd(&dev_avgPixelColor[0], dev_noNoise[i][j + modifier]);
						atomicAdd(&dev_avgPixelColorCounter[0], 1);
						__syncthreads();
					}
				}
			}
		}
	}
	__syncthreads();
	if (threadIdx.x == 0)
	{
		dev_avgPixelColor[0] = __float2int_rn(dev_avgPixelColor[0] / dev_avgPixelColorCounter[0]);
	}
}

void AVGPixelColor_CPU()
{
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < M; j++)
		{
			if (dev_noNoise[i][j] <= 205)
			{
				avgPixelColor[0] += noNoise[i][j];
				avgPixelColorCounter[0]++;
			}
		}
	}

	avgPixelColor[0] = round(avgPixelColor[0] / avgPixelColorCounter[0]);
}

//Konvert�l�s 0-ra �s 255-re

__global__ void ConvertToBlackAndWhite()
{
	int j = threadIdx.x;
	if (dev_M <= dev_MaxThreads)
	{
		if (j < dev_M)
		{
			for (int i = 0; i < dev_N; i++)
			{
				if (dev_noNoise[i][j] <= dev_avgPixelColor[0])
				{
					dev_blackAndWhite[i][j] = 0;
				}
				else
				{
					dev_blackAndWhite[i][j] = 255;
				}
			}
		}
	}
	else
	{
		int run = __float2int_ru(dev_M / dev_MaxThreads);
		for (int block_i = 0; block_i < run; block_i++)
		{
			int modifier = block_i * dev_MaxThreads;
			if (j + modifier < dev_M)
			{
				for (int i = 0; i < dev_N; i++)
				{
					if (dev_noNoise[i][j + modifier] <= dev_avgPixelColor[0])
					{
						dev_blackAndWhite[i][j + modifier] = 0;
					}
					else
					{
						dev_blackAndWhite[i][j + modifier] = 255;
					}
				}
			}
		}
	}
}

void ConvertToBlackAndWhite_CPU()
{
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < M; j++)
		{
			if (noNoise[i][j] <= avgPixelColor[0])
			{
				blackAndWhite[i][j] = 0;
			}
			else
			{
				blackAndWhite[i][j] = 255;
			}
		}
	}
}


__global__ void ConvertToValueMatrix()
{
	int j = threadIdx.x;
	if (dev_M <= dev_MaxThreads)
	{
		if (j < dev_M)
		{
			for (int i = 0; i < dev_N; i++)
			{
				if (dev_blackAndWhite[i][j] == 0)
				{
					dev_valueMatrix[i][j] = -3;
				}
				else
				{
					dev_valueMatrix[i][j] = 0;
				}
			}
		}
	}
	else
	{
		int run = __float2int_ru(dev_M / dev_MaxThreads);
		for (int block_i = 0; block_i < run; block_i++)
		{
			int modifier = block_i * dev_MaxThreads;
			if (j + modifier < dev_M)
			{
				for (int i = 0; i < dev_N; i++)
				{
					if (dev_blackAndWhite[i][j + modifier] == 0)
					{
						dev_valueMatrix[i][j + modifier] = -3;
					}
					else
					{
						dev_valueMatrix[i][j + modifier] = 0;
					}
				}
			}
		}
	}
}

void ConvertToValueMatrix_CPU()
{
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < M; j++)
		{
			if (blackAndWhite[i][j] == 0)
			{
				valueMatrix[i][j] = -3;
			}
			else
			{
				valueMatrix[i][j] = 0;
			}
		}
	}
}

void Custom_printf(float arr[N][M])
{
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < M; j++)
		{
			printf("%003.0f	", arr[i][j]);
		}
		printf("\n");
	}
	printf("\n");
}

void Custom_printf(int arr[N][M])
{
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < M; j++)
		{
			printf("%003d	", arr[i][j]);
		}
		printf("\n");
	}
	printf("\n");
}

void Custom_printf(int vector[NM])
{
	for (int i = 0; i < NM; i++)
	{
		printf("%003d	", vector[i]);
	}
	printf("\n\n");
}

void Custom_printf(int number)
{
	printf("%003d", number);
	printf("\n\n");
}

void Custom_printf(char text[])
{
	printf("%s\n", text);
}

void Custom_printf(char text[], int number)
{
	printf("%s %d\n", text, number);
}

void Custom_printf(char text[], float number)
{
	printf("%s %f\n", text, number);
}

void Custom_printf(bool isCPU, char functionName[])
{
	if (isCPU)
	{
		printf("[CPU] - %s\n", functionName);
	}
	else
	{
		printf("|GPU| - %s\n", functionName);
	}
}

void Custom_printf(bool isCPU, char functionName[], float time)
{
	if (isCPU)
	{
		printf("[CPU] - %s, time: %f millisec\n", functionName, time);
	}
	else
	{
		printf("|GPU| - %s, time: %f millisec\n", functionName, time);
	}
}


void ClockStart()
{
	hipEventRecord(start);
}

void ClockStop(float& milliseconds)
{
	milliseconds = 0;
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
}


int main()
{

	float v_threshold = 10;

	Mat img = imread("C:\\Users\\loahc\\Documents\\GitHub\\Thesis_HJC885\\Unity\\Thesis_HJC885\\Assets\\screenshots\\testpic\\screen_1024x768_2019-11-25_15-22-04_01.png");
	Mat img2 = imread("C:\\Users\\loahc\\Documents\\GitHub\\Thesis_HJC885\\Unity\\Thesis_HJC885\\Assets\\screenshots\\testpic\\screen_1024x768_2019-11-25_15-22-04_02.png");
	
	Custom_printf("Picture Size N: ", N);
	Custom_printf("Picture Size M: ", M);

	namedWindow("pic_1", WINDOW_NORMAL);
	resizeWindow("pic_1", img.cols, img.rows);
	imshow("pic_1", img);

	namedWindow("newpic", WINDOW_NORMAL);
	resizeWindow("newpic", img2.cols, img2.rows);
	imshow("newpic", img2);




	int height = img.rows;
	int width = img.cols;

	Mat imgadd = imread("C:\\Users\\loahc\\Documents\\GitHub\\Thesis_HJC885\\Unity\\Thesis_HJC885\\Assets\\screenshots\\testpic\\screen_1024x768_2019-11-25_15-22-04_01.png");
	Mat img2add = imread("C:\\Users\\loahc\\Documents\\GitHub\\Thesis_HJC885\\Unity\\Thesis_HJC885\\Assets\\screenshots\\testpic\\screen_1024x768_2019-11-25_15-22-04_02.png");

	//convert image from CV::MAT to float*.
	Mat dstadd;
	imgadd.convertTo(dstadd, CV_32F);
	float* dataadd = dstadd.ptr<float>();

	Mat dst2add;
	img2add.convertTo(dst2add, CV_32F);
	float* data2add = dst2add.ptr<float>();


	//ADDED

	for (int i = 0; i < (height * width) * 3; i += 3)
	{
		float r = (dataadd[i] + data2add[i]) / 2;
		float g = (dataadd[i + 1] + data2add[i + 1]) / 2;
		float b = (dataadd[i + 2] + data2add[i + 2]) / 2;


		dataadd[i] = r;
		dataadd[i + 1] = g;
		dataadd[i + 2] = b;
	}


	//convert back the image from float* to CV::MAT.
	Mat destadd(height, width, CV_32FC3, dataadd);

	//print the image
	imwrite("addpic.jpg", destadd);
	Mat img3add = imread("addpic.jpg");
	namedWindow("AddedPic", WINDOW_NORMAL);
	resizeWindow("AddedPic", img3add.cols, img3add.rows);
	imshow("AddedPic", img3add);


	//check whether image loaded is empty or not.
	if (img.empty())
	{
		cerr << "no image"; return -1;
	}

	//convert image from CV::MAT to float*.
	Mat dst;
	img.convertTo(dst, CV_32F);
	float* data = dst.ptr<float>();

	Mat dst2;
	img2.convertTo(dst2, CV_32F);
	float* data2 = dst2.ptr<float>();


		//sub

	for (int i = 0; i < (height * width) * 3; i += 3)
	{
		float r = abs((data[i] - data2[i]) / 2);
		float g = abs((data[i + 1] - data2[i + 1]) / 2);
		float b = abs((data[i + 2] - data2[i + 2]) / 2);

		//float avg = (r + g + b) / 3;
		data[i] = r;
		data[i + 1] = g;
		data[i + 2] = b;
	}


	//convert back the image from float* to CV::MAT.
	Mat dest(height, width, CV_32FC3, data);

	//print the image
	imwrite("subpic.jpg", dest);
	Mat img3 = imread("subpic.jpg");
	namedWindow("subpic", WINDOW_NORMAL);
	resizeWindow("subpic", img3.cols, img3.rows);
	imshow("subpic", img3);



		//Makewhite and black only
	for (int i = 0; i < (height * width) * 3; i += 3)
	{

		float r = data[i];
		float g = data[i + 1];
		float b = data[i + 2];
		if (r >= v_threshold || g >= v_threshold || b >= v_threshold)
		{
			data[i] = 255;
			data[i + 1] = 255;
			data[i + 2] = 255;
		}


	}
	Mat dest2(height, width, CV_32FC3, data);

	//print the image
	imwrite("blackandwhite.jpg", dest2);
	//Mat img5 = imread("blackandwhite.png");
	Mat img4 = imread("blackandwhite.jpg");

	Mat src, src_gray;

	/// Read the image
	src = imread("blackandwhite.jpg", 1);

	if (!src.data)
	{
		return -1;
	}

	/// Convert it to gray
	cvtColor(src, src_gray, COLOR_RGB2GRAY);
	namedWindow("Hough Circle Transform Gray", WINDOW_NORMAL);
	imshow("Hough Circle Transform Gray", src);

	/// Reduce the noise so we avoid false circle detection
	GaussianBlur(src_gray, src_gray, Size(9, 9), 2, 2);
	namedWindow("Hough Circle Transform Gauss", WINDOW_NORMAL);
	imshow("Hough Circle Transform Gauss", src);

	vector<Vec3f> circles;
	for (int maxR = 10; maxR < 200; maxR = maxR + 200-9)
	{

		/// Apply the Hough Transform to find the circles
		//		WorkingHoughCircles(src_gray, circles, HOUGH_GRADIENT, 1, src_gray.rows / 100, 10, 10, 0, 100);
		HoughCircles(src_gray, circles, HOUGH_GRADIENT, 1, src_gray.rows / 100, 10, 10, 0, 0);

		/// Draw the circles detected
		for (size_t i = 0; i < circles.size(); i++)
		{
			Point center(cvRound(circles[i][0]), cvRound(circles[i][1]));
			int radius = cvRound(circles[i][2]);
			// circle center
			circle(src, center, 2, Scalar(0, 255, 0),2, 8, 0);
			// circle outline
			circle(src, center, radius, Scalar(0, 0, 255), 2, 8, 0);
			if (i > 0)
			{
				Point center1(cvRound(circles[i - 1][0]), cvRound(circles[i - 1][1]));
				printf("Point1 <%i, \t", center1.x);
				printf("%i >\n", center1.y);

				Point center2(cvRound(circles[i][0]), cvRound(circles[i][1]));

				printf("Point2 <%i, \t", center2.x);
				printf("%i >\n", center2.y);

				line(src, center1, center2, Scalar(0, 0, 255), 2, 8, 0);
			}
		}
	}

	/// Show your results
	namedWindow("Hough Circle Transform", WINDOW_NORMAL);
	imshow("Hough Circle Transform", src);



	Mat merged = imread("C:\\Users\\loahc\\Documents\\GitHub\\Thesis_HJC885\\Unity\\Thesis_HJC885\\Assets\\screenshots\\testpic\\screen_1024x768_2019-11-25_15-22-04_01.png");
	Mat imgsecond = imread("C:\\Users\\loahc\\Documents\\GitHub\\Thesis_HJC885\\Unity\\Thesis_HJC885\\Assets\\screenshots\\testpic\\screen_1024x768_2019-11-25_15-22-04_02.png");

	Mat dstmerged;
	merged.convertTo(dstmerged, CV_32F);
	float* datamerged = dstmerged.ptr<float>();

	Mat dstsecond;
	imgsecond.convertTo(dstsecond, CV_32F);
	float* data2second = dstsecond.ptr<float>();

	Mat dstcircles;
	src.convertTo(dstcircles, CV_32F);
	float* datamcircles = dstcircles.ptr<float>();

	//add

	for (int i = 0; i < (height * width) * 3; i += 3)
	{
		float r = (datamerged[i] + data2second[i] + datamcircles[i]) / 3;
		float g = (datamerged[i+1] + data2second[i + 1] + datamcircles[i + 1] )/ 3;
		float b = (datamerged[i + 2] + data2second[i + 2] + datamcircles[i + 2]) / 3;

		//float avg = (r + g + b) / 3;
		datamerged[i] = r;
		datamerged[i + 1] = g;
		datamerged[i + 2] = b;
	}




	//////////////////////////////////////////////INNEN START////////////////////////////////////////////////////////////
	hipEventCreate(&start);
	hipEventCreate(&stop);


	float milliseconds = 0;



	Custom_printf("");
	Custom_printf("");

	MEASURE_TIME(1, "LoadPicture_CPU", RandomPicture_CPU());

	//MEASURE_TIME(1, "LoadedPicture_CPU", LoadPicture_CPU(img));





	ClockStart();
	hipMemcpyToSymbol(HIP_SYMBOL(dev_channel_r), channel_r, N * M * sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(dev_channel_g), channel_g, N * M * sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(dev_channel_b), channel_b, N * M * sizeof(float));
	ClockStop(milliseconds);
	Custom_printf(false, "Copy", milliseconds);

	ClockStart();
	GrayScale << < 1, MaxThreads >> > ();
	ClockStop(milliseconds);
	Custom_printf(false, "Grayscale", milliseconds);




	ConvertArrayToVector << < 1, MaxThreads >> > ();


	ClockStart();
	MinSearch << < 1, MaxThreads >> > ();

	ClockStop(milliseconds);
	Custom_printf(false, "Minimum Search", milliseconds);



	ClockStart();
	MaxSearch << < 1, MaxThreads >> > ();

	ClockStop(milliseconds);
	Custom_printf(false, "Maximum Search", milliseconds);


	ClockStart();
	DarkPixelNumber << <1, MaxThreads >> > ();

	ClockStop(milliseconds);
	Custom_printf(false, "Dark Pixel Counter", milliseconds);

	hipMemcpyFromSymbol(res_darkPixelCounter, HIP_SYMBOL(dev_darkPixelCounter), 1 * sizeof(int));

	if (NM / 2 < res_darkPixelCounter[0])
	{
		ClockStart();
		ColorInverter << <1, MaxThreads >> > ();

		ClockStop(milliseconds);
		Custom_printf(false, "Color Inverter", milliseconds);
	}



	ClockStart();
	GetGaussValue << <1, GaussSize* GaussSize >> > ();

	ClockStop(milliseconds);
	Custom_printf(false, "Gauss Value", milliseconds);



	ClockStart();
	GaussTransformation << <1, MaxThreads >> > ();

	ClockStop(milliseconds);
	Custom_printf(false, "Gauss Transformation", milliseconds);



	ClockStart();
	AVGPixelColor << <1, MaxThreads >> > ();

	ClockStop(milliseconds);
	Custom_printf(false, "AVG Pixel Color", milliseconds);



	ClockStart();
	ConvertToBlackAndWhite << <1, MaxThreads >> > ();

	ClockStop(milliseconds);
	Custom_printf(false, "Convert To Black And White", milliseconds);



	ClockStart();
	ConvertToValueMatrix << <1, MaxThreads >> > ();

	ClockStop(milliseconds);
	Custom_printf(false, "Convert To Value Matrix", milliseconds);



	Custom_printf("");
	Custom_printf("All GPU Process Finished");
	Custom_printf("");



	ClockStart();
	hipMemcpyFromSymbol(res_grayScale, HIP_SYMBOL(dev_grayScale), N * M * sizeof(int));
	hipMemcpyFromSymbol(res_forMinMaxSearch, HIP_SYMBOL(dev_forMinMaxSearch), N * M * sizeof(int));
	hipMemcpyFromSymbol(res_globalMin, HIP_SYMBOL(dev_globalMin), 1 * sizeof(int));
	hipMemcpyFromSymbol(res_globalMax, HIP_SYMBOL(dev_globalMax), 1 * sizeof(int));
	hipMemcpyFromSymbol(res_histogram, HIP_SYMBOL(dev_histogram), N * M * sizeof(int));
	hipMemcpyFromSymbol(res_GaussValue, HIP_SYMBOL(dev_GaussValue), 1 * sizeof(int));
	hipMemcpyFromSymbol(res_noNoise, HIP_SYMBOL(dev_noNoise), N * M * sizeof(int));
	hipMemcpyFromSymbol(res_avgPixelColor, HIP_SYMBOL(dev_avgPixelColor), 1 * sizeof(int));
	hipMemcpyFromSymbol(res_blackAndWhite, HIP_SYMBOL(dev_blackAndWhite), N * M * sizeof(int));
	hipMemcpyFromSymbol(res_valueMatrix, HIP_SYMBOL(dev_valueMatrix), N * M * sizeof(int));

	ClockStop(milliseconds);
	Custom_printf(false, "All Value Copy Back", milliseconds);


	MEASURE_TIME(1, "GrayScale_CPU", GrayScale_CPU());
	MEASURE_TIME(1, "MinSearch_CPU", MinSearch_CPU());
	MEASURE_TIME(1, "MaxSearch_CPU", MaxSearch_CPU());
	MEASURE_TIME(1, "DarkPixelNumber_CPU", DarkPixelNumber_CPU());


	MEASURE_TIME(1, "GetGaussValue_CPU", GetGaussValue_CPU());
	MEASURE_TIME(1, "GaussTransformation_CPU", GaussTransformation_CPU());
	MEASURE_TIME(1, "AVGPixelColor_CPU", AVGPixelColor_CPU());
	MEASURE_TIME(1, "ConvertToBlackAndWhite_CPU", ConvertToBlackAndWhite_CPU());

	//Custom_printf(channel_r);
	//Custom_printf(channel_g);
	//Custom_printf(channel_b);

	
	//Custom_printf(channel_r);

	//Custom_printf(channel_g);

	//Custom_printf(channel_b);

	//Custom_printf(grayScale);

	//Custom_printf(forMinMaxSearch);

	//Custom_printf(globalMin[0]);

	//Custom_printf(globalMax[0]);

	//Custom_printf(histogram);

	//Custom_printf(GaussValue[0]);

	//Custom_printf(noNoise);

	//Custom_printf(avgPixelColor[0]);

	//Custom_printf(blackAndWhite);

	//Custom_printf(valueMatrix);

	
	waitKey(0);
	return 0;
}